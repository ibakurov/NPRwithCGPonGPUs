
#include <hip/hip_runtime.h>
/** =====================================Stack related definitions==================================== */
/** The size of the interpreter stack */
#define STACK_SIZE 128
#define push(A) do { sp++;stack[sp]=A; if(sp >= STACK_SIZE) printf("Stack overflow\n");} while(false)
#define pop(A) do{ A=stack[sp];sp--; if(sp < -1) printf("Stack underflow\n");} while(false)
/** ================================================================================================== */

/** The number of pixels, which we need to paint */
#define BLOCK_SIZE 512	// Used for the shared memory definitions

/************************************************************************************************************
 ************************************************************************************************************/

//TODO DOC: sadly there is only support for 1 pitch value for all input instances (which should be more than enough)
extern "C"
__global__ void evaluate(int* x, int* y,
	 												float* redInput, float* greenInput, float* blueInput,
													float* redCanvas, float* greenCanvas, float* blueCanvas,
													float* opacity,
	 												int* exp1, int* exp2, int* exp3,
													float* luminance, float* ERC,
													float* mean5x5, float* mean7x7, float* mean9x9, float* mean11x11, float* mean13x13,
													float* std5x5, float* std7x7, float* std9x9, float* std11x11, float* std13x13,
													float* min5x5, float* min7x7, float* min9x9, float* min11x11, float* min13x13,
													float* max5x5, float* max7x7, float* max9x9, float* max11x11, float* max13x13,
													int inputPitch, float* output, int outputPitch,
												/*const char** __restrict__ individuals,*/ const int pixelsPerBlockCount, const int pixelsCount/*, const int maxLength*/) {

	//In CUDA we have grids, which are blocks of threads!
	//Thus, blockIndex, goes from 0 to N-1 (Depends on what we instantiated it to be)
	//blockIdx.x is simply the index of the block in the grid
	int blockIndex = blockIdx.x;
	//threadIdx.x is the index of the thread inside the block
	int threadIndex = threadIdx.x;
	//blockDim.x is the number of threads per block
	int blockDimension = blockDim.x;

  //Check if this blockIndex is in the range of active pixels, we are working with
	if (blockIndex >= pixelsPerBlockCount)
		return;

	// Thread to data index mapping.
	int tid = blockIndex * blockDimension + threadIndex;

  //Check if our index is out of problem's size, thus there is not going to be data for this index.
	if (tid >= pixelsCount)
		return;

	//Also if this pixel is just white, we don't do anything with it
	if (opacity[tid] == 255) {
		output[(tid*3)] = -1;
		output[(tid*3) + 1] = -1;
		output[(tid*3) + 2] = -1;
		return;
	}

	float stack[STACK_SIZE];	// The stack is defined as the same type as the kernel output
	int sp;

		float rgb [3];

		for (int k = 0; k < 3; k++) {

				// Reset the stack pointer
				sp = - 1;

				//Get a proper expression to work with
				int* expression;
				if (k == 0) {
					expression = exp1;
				} else if (k == 1) {
					expression = exp2;
				} else {
					expression = exp3;
				}

				//This is used, in case if you want to comment out some of the inputs, so that we don't need to go and change all cases for functions.
				const int totalNumberOfTerminals = 31;

				int l = 0;	// Maintains the current index in the expression
				while(expression[l] != 0)
				{
					switch(expression[l])
					{
						case 1: {
							// printf("X (%i)", x[tid]);
							push(x[tid]);
						} break;
						case 2: {
							// printf("Y (%i)", x[tid]);
							push(y[tid]);
						} break;
						case 3: {
							// printf("redInput (%f)\n", redInput[tid]/ 255.f);
							push(redInput[tid]/ 255.f);
						} break;
						case 4: {
							// printf("greenInput (%i)", greenInput[tid]);
							push(greenInput[tid]/ 255.0);
						} break;
						case 5: {
							// printf("blueInput (%i)", blueInput[tid]);
							push(blueInput[tid]/ 255.0);
						} break;
						case 6: {
							// printf("redCanvas (%i)", redCanvas[tid]);
							push(redCanvas[tid]/ 255.0);
						} break;
						case 7: {
							// printf("greenCanvas (%i)", greenCanvas[tid]);
							push(greenCanvas[tid]/ 255.0);
						} break;
						case 8: {
							// printf("blueCanvas (%i)", blueCanvas[tid]);
							push(blueCanvas[tid]/ 255.0);
						} break;
						case 9: {
							// printf("opacity (%i)", opacity[tid]);
							push(opacity[tid]);
						} break;
						case 10: {
							// printf("luminance (%f)", luminance[tid]);
							push(luminance[tid]);
						} break;
						case 11: {
							// printf("ERC (%f)", ERC[tid]);
							push(ERC[tid]);
						} break;
						case 12: {
							// printf("mean5x5 (%f)", mean5x5[tid]);
							push(mean5x5[tid]);
						} break;
						case 13: {
							// printf("mean7x7 (%f)", mean7x7[tid]);
							push(mean7x7[tid]);
						} break;
						case 14: {
							// printf("mean9x9 (%f)", mean9x9[tid]);
							push(mean9x9[tid]);
						} break;
						case 15: {
							// printf("mean11x11 (%f)", mean11x11[tid]);
							push(mean11x11[tid]);
						} break;
						case 16: {
							// printf("mean13x13 (%f)", mean13x13[tid]);
							push(mean13x13[tid]);
						} break;
						case 17: {
							// printf("std5x5 (%f)", std5x5[tid]);
							push(std5x5[tid]);
						} break;
						case 18: {
							// printf("std7x7 (%f)", std7x7[tid]);
							push(std7x7[tid]);
						} break;
						case 19: {
							// printf("std9x9 (%f)", std9x9[tid]);
							push(std9x9[tid]);
						} break;
						case 20: {
							// printf("std11x11 (%f)", std11x11[tid]);
							push(std11x11[tid]);
						} break;
						case 21: {
							// printf("std13x13 (%f)", std13x13[tid]);
							push(std13x13[tid]);
						} break;
						case 22: {
							// printf("min5x5 (%f)", min5x5[tid]);
							push(min5x5[tid]);
						} break;
						case 23: {
							// printf("min7x7 (%f)", min7x7[tid]);
							push(min7x7[tid]);
						} break;
						case 24: {
							// printf("min9x9 (%f)", min9x9[tid]);
							push(min9x9[tid]);
						} break;
						case 25: {
							// printf("min11x11 (%f)", min11x11[tid]);
							push(min11x11[tid]);
						} break;
						case 26: {
							// printf("min13x13 (%f)", min13x13[tid]);
							push(min13x13[tid]);
						} break;
						case 27: {
							// printf("max5x5 (%f)", max5x5[tid]);
							push(max5x5[tid]);
						} break;
						case 28: {
							// printf("max7x7 (%f)", max7x7[tid]);
							push(max7x7[tid]);
						} break;
						case 29: {
							// printf("max9x9 (%f)", max9x9[tid]);
							push(max9x9[tid]);
						} break;
						case 30: {
							// printf("max11x11 (%f)", max11x11[tid]);
							push(max11x11[tid]);
						} break;
						case 31: {
							// printf("max13x13 (%f)", max13x13[tid]);
							push(max13x13[tid]);
						} break;
						case (totalNumberOfTerminals + 1): {
							// printf("+ ");
							float second;pop(second);
							float first;pop(first);
							float final = second + first;
							push(final);
						} break;
						case (totalNumberOfTerminals + 2): {
							// printf("- ");
							float second;pop(second);
							float first;pop(first);
							float final = second - first;
							push(final);
						} break;
						case (totalNumberOfTerminals + 3): {
							// printf("* ");
							float second;pop(second);
							float first;pop(first);
							float final = second * first;
							push(final);
						} break;
						case (totalNumberOfTerminals + 4): {
							// printf("/ ");
							float second;pop(second);
							float first;pop(first);
							if (second == 0) {
								float final = 1.0;
								push(final);
							} else {
								float final = second / first;
								push(final);
							}
						} break;
						case (totalNumberOfTerminals + 5): {
							// printf("neg ");
							float first;pop(first);
							float final = 0 - first;
							push(final);
						} break;
						case (totalNumberOfTerminals + 6): {
							// printf("sin ");
							float first;pop(first);
							float final = sinf(first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 7): {
							// printf("cos ");
							float first;pop(first);
							float final = cosf(first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 8): {
							// printf("iflez ");
							float fourth;pop(fourth);
							float third;pop(third);
							float second;pop(second);
							float first;pop(first);
							if (fourth <= third) {
								push(second);
							} else {
								push(first);
							}
						} break;
						case (totalNumberOfTerminals + 9): {
							// printf("abs ");
							float first;pop(first);
							float final = fabs(first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 10): {
							// printf("round ");
							float first;pop(first);
							float final = round(first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 11): {
							// printf("avg ");
							float second;pop(second);
							float first;pop(first);
							float final = (second + first) / 2;
							push(final);
						} break;
						case (totalNumberOfTerminals + 12): {
							// printf("log ");
							float first;pop(first);
							float final;
							if (first == 0) {
								final = 0.f;
							} else {
								final = log(fabs(first));
							}
							push(final);
						} break;
						case (totalNumberOfTerminals + 13): {
							// printf("exp ");
							float first;pop(first);
							float final = exp(fmodf(first, 10));
							push(final);
						} break;
						case (totalNumberOfTerminals + 14): {
							// printf("min ");
							float second;pop(second);
							float first;pop(first);
							float final = fminf(second, first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 15): {
							// printf("max ");
							float second;pop(second);
							float first;pop(first);
							float final = fmaxf(second, first);
							push(final);
						} break;
						case (totalNumberOfTerminals + 16): {
							// printf("brt ");
							float value;pop(value);
							if (value < 0) { value = 0; }
							if (value > 1) { value = 1; }
							float factor;pop(factor);
							if (factor > 1) { factor = factor - (int)factor; }
							float final = (((value * 255.0) * (1 - factor) / 255.0 + factor) * 255.0) / 255.0;
							push(final);
						} break;
						case (totalNumberOfTerminals + 17): {
							// printf("drk ");
							float factor;pop(factor);
							float value;pop(value);
							if (value < 0) { value = 0; }
							if (value > 1) { value = 1; }
							if (factor > 1) { factor = factor - (int)factor; }
							float final = (((value * 255.0) * (1 - factor) / 255.0) * 255.0) / 255.0;
							push(final);
						} break;
						case (totalNumberOfTerminals + 18): {
							// printf("brn ");
							float opacityValue;pop(opacityValue);
							float value2;pop(value2);
							float value1;pop(value1);
							if (value1 < 0) { value1 = 0.f; }
							if (value1 > 1) { value1 = 1.f; }
							if (value2 < 0) { value2 = 0.0f; }
							if (value2 > 1) { value2 = 1.0f; }
							opacityValue = fabs(opacityValue);
							if (opacityValue == 0) { opacityValue = 0.1f; }
							if (opacityValue > 0.1) { opacityValue = 0.1f / opacityValue; }
							value1 = value1 * 255.0f;
							value2 = value2 * 255.0f;
							if (value2 == 0) { value2 = 1.0f; }
							float final = 255.0f - (255.0f-value1)*(1.0f + 254.0f*(255.0f/value2)/255.0f);
							final = (value1 * (1.0f - opacityValue) + final * opacityValue);
							if (final>255)	{ final = 255.0f; }
							final = final / 255.0f;
							push(final);
						} break;
						case (totalNumberOfTerminals + 19): {
							// printf("dgn ");
							float opacityValue;pop(opacityValue);
							float value2;pop(value2);
							float value1;pop(value1);
							if (value1 < 0) { value1 = 0; }
							if (value1 > 1) { value1 = 1; }
							if (value2 < 0) { value2 = 0; }
							if (value2 > 1) { value2 = 1; }
							opacityValue = fabs(opacityValue);
							if (opacityValue == 0) { opacityValue = 0.1; }
							if (opacityValue > 0.1) { opacityValue = 0.1 / opacityValue; }
							value1 = value1 * 255;
							value2 = value2 * 255;
							float final = ( value1 * (1 - opacityValue) + value2 * opacityValue);
							if (final>255)	{ final = 255; }
							final = final / 255.0;
							push(final);
						} break;
						case (totalNumberOfTerminals + 20): {
							// printf("nbld ");
							float opacityValue;pop(opacityValue);
							float value2;pop(value2);
							float value1;pop(value1);
							if (value1 < 0) { value1 = 0; }
							if (value1 > 1) { value1 = 1; }
							if (value2 < 0) { value2 = 0; }
							if (value2 > 1) { value2 = 1; }
							opacityValue = fabs(opacityValue);
							if (opacityValue == 0) { opacityValue = 0.1; }
							if (opacityValue > 0.1) { opacityValue = 0.1 / opacityValue; }
							value1 = value1 * 255;
							value2 = value2 * 255;
							float final = ( value1 * (1 - opacityValue) + value2 * opacityValue);
							if (final>255)	{ final = 255; }
							final = final / 255.0;
							push(final);
						} break;
						case (totalNumberOfTerminals + 21): {
							// printf("dbld ");
							float opacityValue;pop(opacityValue);
							float value2;pop(value2);
							float value1;pop(value1);
							if (value1 < 0) { value1 = 0; }
							if (value1 > 1) { value1 = 1; }
							if (value2 < 0) { value2 = 0; }
							if (value2 > 1) { value2 = 1; }
							opacityValue = fabs(opacityValue);
							if (opacityValue == 0) { opacityValue = 0.1; }
							if (opacityValue > 0.1) { opacityValue = 0.1 / opacityValue; }
							value1 = value1 * 255;
							value2 = value2 * 255;
							float final = ( (fabs(value1 - value2) * opacityValue)  + ( value2 * (1 - opacityValue)));
							if (final>255)	{ final = 255; }
							else if (final<0)	{ final = 0; }
							final = final / 255.0;
							push(final);
						} break;
						case (totalNumberOfTerminals + 22): {
							// printf("obld ");
							float value2;pop(value2);
							float value1;pop(value1);
							if (value1 < 0) { value1 = 0; }
							if (value1 > 1) { value1 = 1; }
							if (value2 < 0) { value2 = 0; }
							if (value2 > 1) { value2 = 1; }
							value1 = value1 * 255;
							value2 = value2 * 255;
							float final;
							if (value1 > 128) { final = 255 - (255- value1)*(255-value2)/128; }
        			else { final = value1*value2 / 128; }
        			if (final>255)	{ final = 255; }
        			else if (final<0)	{ final = 0; }
							final = final / 255.0;
							push(final);
						} break;
						default:printf("Unrecognized OPCODE in the expression tree!");break;
					}
					// printf("sp: %d", sp);
					l++;
				}

				if (l == 0) {
					printf("Expression is empty %i!\n", tid);
					rgb[k] = 255;
				} else {
						// Pop the top of the stack
						float stackTop;
						pop(stackTop);

						if(sp!=-1) {
							printf("Stack pointer is not -1 but is %d\n", sp);
						}

						rgb[k] = stackTop;
				}
		}

		// Assign the results to outputs
		output[(tid*3)] = rgb[0];
		output[(tid*3) + 1] = rgb[1];
		output[(tid*3) + 2] = rgb[2];
}
